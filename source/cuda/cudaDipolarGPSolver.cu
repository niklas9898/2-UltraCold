#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
*
*    This file is part of the UltraCold project.
*
*    UltraCold is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    any later version.
*    UltraCold is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*    You should have received a copy of the GNU General Public License
*    along with UltraCold.  If not, see <https://www.gnu.org/licenses/>.
*
*--------------------------------------------------------------------------------*/

#include <random>
#include "cudaDipolarGPSolver.cuh"
#include "mesh_fourier_space.hpp"
#include "DataWriter.hpp"
#include "hipcub/hipcub.hpp"
#include "simple_kernels.cuh"
#include "solver_kernels.cuh"
#include "DFtCalculator.hpp"
#include "boost/math/special_functions/bessel.hpp"

#define PI 3.1415926535897932384626433
#define TWOPI (2*PI)

namespace UltraCold
{
    namespace cudaSolvers
    {

        /**
         * @brief Constructor for 2d problems
         */

        DipolarGPSolver::DipolarGPSolver(Vector<double> &x,
                                         Vector<double> &y,
                                         Vector<std::complex<double>> &psi_0,
                                         Vector<double> &Vext,
                                         double scattering_length,
                                         double dipolar_length,
                                         double alpha)
        {

            // Check the order and extent of the Vectors provided
            assert(x.order()==1);
            assert(y.order()==1);
            assert(psi_0.order() == 2);
            assert(Vext.order() == 2);
            nx=x.extent(0);
            ny=y.extent(0);
            assert(psi_0.extent(0) == nx);
            assert(psi_0.extent(1) == ny);
            assert(Vext.extent(0) == nx);
            assert(Vext.extent(1) == ny);
            problem_is_2d=true;
            npoints=nx*ny;

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,npoints*sizeof(double));
            hipMalloc(&kmod2_d,             npoints*sizeof(double));
            hipMalloc(&density_d,           npoints*sizeof(double));
            hipMalloc(&wave_function_d,     npoints*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              npoints*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  npoints*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> ky(ny);
            Vector<double> kmod2(nx,ny);
            create_mesh_in_Fourier_space(x,y,kx,ky);
            for (size_t i = 0; i < nx; ++i)
                for (size_t j = 0; j < ny; ++j)
                    kmod2(i,j) = std::pow(kx(i),2) +
                                 std::pow(ky(j),2);
            hipMemcpy(kmod2_d,kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dy = y(1)-y(0);
            dv = dx*dy;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx,ny);

            // Initialize the host vectors containing the mesh axis. This can be useful in particular for data output
            x_axis.reinit(nx);
            y_axis.reinit(ny);
            x_axis=x;
            y_axis=y;
            kx_axis.reinit(nx);
            ky_axis.reinit(ny);
            kx_axis=kx;
            ky_axis=ky;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMalloc(&y_axis_d,ny*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(y_axis_d,y_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMalloc(&ky_axis_d,ny*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(ky_axis_d,ky_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx,ny);
            for(int i = 0; i < nx; ++i)
                for(int j = 0; j < ny; ++j)
                    r2mod(i,j) = std::pow(x(i),2)+std::pow(y(j),2);
            hipMalloc(&r2mod_d,npoints*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize the Fourier transform of the dipolar potential

            double epsilon_dd = 0.0;
            if(scattering_length != 0)
                epsilon_dd = dipolar_length/scattering_length;

            Vtilde.reinit(nx,ny);
            for (int i = 0; i < nx; ++i)
                for (int j = 0; j < ny; ++j)
                {
                    double qd = TWOPI * kx[i] / sqrt(2);
                    double q  = TWOPI * sqrt(pow(kx[i], 2) + pow(ky[j], 2)) / sqrt(2);
                    double value =
                            sqrt(8*PI) * scattering_length * epsilon_dd *
                            (
                                    (-1 + 3*sqrt(PI) * pow(qd,2)/q * exp(pow(q,2)) * erfc(q)) * pow(sin(alpha),2) +
                                    ( 2 - 3*sqrt(PI) * q * exp(pow(q,2)) * erfc(q)) * pow(cos(alpha),2)
                            );
                    if (isnan(value) && kx(i) == 0 && ky(j) == 0)
                        Vtilde(i, j) = sqrt(8*PI)*scattering_length*epsilon_dd*(3*std::pow(std::cos(alpha),2)-1);
                    else if(isnan(value) && kx(i) != 0 && ky(j) != 0)
                        Vtilde(i,j) = 0.0;
                    else
                        Vtilde(i, j) = value;
                }

            hipMalloc(&Vtilde_d,npoints*sizeof(hipDoubleComplex));
            hipMemcpy(Vtilde_d,Vtilde.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMalloc(&Phi_dd_d,npoints*sizeof(hipDoubleComplex));

            // Initialize gamma(\epsilon_dd) for the LHY correction
            hipMallocManaged(&gamma_epsilondd_d,sizeof(double));
            gamma_epsilondd_d[0] = 0.0;

            // Scattering length is divided by sqrt(2PI) here, since in the propagators it is multiplied by 4PI
            scattering_length *= 1./sqrt(2*PI);
            hipMemcpy(scattering_length_d, &scattering_length,1*sizeof(double),hipMemcpyHostToDevice);

            // Initialize the wave function for the output
            wave_function_output.reinit(nx,ny);

        }

        /**
         * @brief Constructor for 3d problems
         */

        DipolarGPSolver::DipolarGPSolver(Vector<double> &x,
                                         Vector<double> &y,
                                         Vector<double> &z,
                                         Vector<std::complex<double>> &psi_0,
                                         Vector<double> &Vext,
                                         double scattering_length,
                                         double dipolar_length,
                                         double theta_mu,
                                         double phi_mu,
                                         bool add_lhy_correction)
        {

            // Check the order and extent of the Vectors provided
            assert(x.order()==1);
            assert(y.order()==1);
            assert(z.order()==1);
            assert(psi_0.order() == 3);
            assert(Vext.order() == 3);
            nx=x.extent(0);
            ny=y.extent(0);
            nz=z.extent(0);
            assert(psi_0.extent(0) == nx);
            assert(psi_0.extent(1) == ny);
            assert(psi_0.extent(2) == nz);
            assert(Vext.extent(0) == nx);
            assert(Vext.extent(1) == ny);
            assert(Vext.extent(2) == nz);
            problem_is_3d=true;
            npoints=nx*ny*nz;

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,npoints*sizeof(double));
            hipMalloc(&kmod2_d,             npoints*sizeof(double));
            hipMalloc(&density_d,           npoints*sizeof(double));
            hipMalloc(&wave_function_d,     npoints*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              npoints*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  npoints*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d, &scattering_length,1      *sizeof(double),         hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> ky(ny);
            Vector<double> kz(nz);
            Vector<double> kmod2(nx,ny,nz);
            create_mesh_in_Fourier_space(x,y,z,kx,ky,kz);
            for (size_t i = 0; i < nx; ++i)
                for (size_t j = 0; j < ny; ++j)
                    for (size_t k = 0; k < nz; ++k)
                        kmod2(i,j,k) = std::pow(kx(i),2)+
                                       std::pow(ky(j),2)+
                                       std::pow(kz(k),2);
            hipMemcpy(kmod2_d, kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dy = y(1)-y(0);
            dz = z(1)-z(0);
            dv = dx*dy*dz;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx,ny,nz);

            // Initialize the host vectors containing the mesh axis. This can be useful in particular for data output
            x_axis.reinit(nx);
            y_axis.reinit(ny);
            z_axis.reinit(nz);
            x_axis=x;
            y_axis=y;
            z_axis=z;
            kx_axis.reinit(nx);
            ky_axis.reinit(ny);
            kz_axis.reinit(nz);
            kx_axis=kx;
            ky_axis=ky;
            kz_axis=kz;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMalloc(&y_axis_d,ny*sizeof(double));
            hipMalloc(&z_axis_d,nz*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(y_axis_d,y_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(z_axis_d,z_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMalloc(&ky_axis_d,ny*sizeof(double));
            hipMalloc(&kz_axis_d,nz*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(ky_axis_d,ky_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(kz_axis_d,kz_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx,ny,nz);
            for(int i = 0; i < nx; ++i)
                for(int j = 0; j < ny; ++j)
                    for(int k = 0; k < nz; ++k)
                        r2mod(i,j,k) = std::pow(x(i),2);
            hipMalloc(&r2mod_d,npoints*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize the Fourier transform of the dipolar potential
            hipMallocManaged(&epsilon_dd_d,sizeof(double));
            epsilon_dd_d[0] = 0.0;
            if(scattering_length != 0)
                epsilon_dd_d[0] = dipolar_length/scattering_length;
            Vtilde.reinit(nx,ny,nz);
            for (int i = 0; i < nx; ++i)
                for (int j = 0; j < ny; ++j)
                    for (int k = 0; k < nz; ++k)
                    {
                        double aux = TWOPI * (
                                kx[i]*sin(theta_mu)*cos(phi_mu) +
                                ky[j]*sin(theta_mu)*sin(phi_mu)+
                                kz[k]*cos(theta_mu));
                        double aux1 = TWOPI * sqrt(pow(kx[i], 2) + pow(ky[j], 2) + pow(kz[k], 2));
                        if (aux1 <= 1.E-6)
                            Vtilde(i,j,k) = -4*PI*scattering_length*epsilon_dd_d[0];
                        else
                            Vtilde(i,j,k) =
                                    12.0 * PI * scattering_length * epsilon_dd_d[0] * (pow(aux/aux1,2)-1.0/3.0);
                    }
            hipMalloc(&Vtilde_d,npoints*sizeof(hipDoubleComplex));
            hipMemcpy(Vtilde_d,Vtilde.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMalloc(&Phi_dd_d,npoints*sizeof(hipDoubleComplex));

            // Initialize gamma(\epsilon_dd) for the LHY correction
            hipMallocManaged(&gamma_epsilondd_d,sizeof(double));
            gamma_epsilondd_d[0] = 0.0;
            if (epsilon_dd_d[0] != 0 && add_lhy_correction)
            {
                gamma_epsilondd_d[0] = 64.0*sqrt(PI)/3*sqrt(pow(scattering_length,5));
                double F_epsilon_dd=0.0;
                int n_theta=1000;
                double d_theta=PI/(n_theta-1);

                std::complex<double> csum;
                std::complex<double> caux;
                csum={0.0,0.0};
                for (int i = 0; i < n_theta; ++i)
                {
                    double theta=i*d_theta;
                    caux = pow(1.0+epsilon_dd_d[0]*(3.0*pow(cos(theta),2)-1.0),5);
                    caux = sqrt(caux);
                    csum += sin(theta)*caux;
                }
                csum *= d_theta;
                F_epsilon_dd = csum.real();
                gamma_epsilondd_d[0] *= F_epsilon_dd;
            }

            // Initialize the wave function for the output
            wave_function_output.reinit(nx,ny,nz);

        }

        /**
         * @brief Constructor for 3d problems including a dipolar cutoff
         */

        DipolarGPSolver::DipolarGPSolver(Vector<double> &x,
                                         Vector<double> &y,
                                         Vector<double> &z,
                                         Vector<std::complex<double>> &psi_0,
                                         Vector<double> &Vext,
                                         double scattering_length,
                                         double dipolar_length,
                                         double theta_mu,
                                         double phi_mu,
                                         Vector<double> dipolar_cutoff,
                                         bool add_lhy_correction)
        {

            // Check the order and extent of the Vectors provided
            assert(x.order()==1);
            assert(y.order()==1);
            assert(z.order()==1);
            assert(psi_0.order() == 3);
            assert(Vext.order() == 3);
            assert(dipolar_cutoff.order() == 1);
            nx=x.extent(0);
            ny=y.extent(0);
            nz=z.extent(0);
            assert(psi_0.extent(0) == nx);
            assert(psi_0.extent(1) == ny);
            assert(psi_0.extent(2) == nz);
            assert(Vext.extent(0) == nx);
            assert(Vext.extent(1) == ny);
            assert(Vext.extent(2) == nz);
            assert(dipolar_cutoff.extent(0) == 3);
            problem_is_3d=true;
            npoints=nx*ny*nz;

            // Initialize the thread grid, i.e. choose the number of cuda threads per block and the number of blocks.
            blockSize = 512;
            gridSize = (npoints + blockSize - 1) / blockSize;

            // Allocate memory for all device arrays
            hipMalloc(&external_potential_d,npoints*sizeof(double));
            hipMalloc(&kmod2_d,             npoints*sizeof(double));
            hipMalloc(&density_d,           npoints*sizeof(double));
            hipMalloc(&wave_function_d,     npoints*sizeof(hipDoubleComplex));
            hipMalloc(&hpsi_d,              npoints*sizeof(hipDoubleComplex));
            hipMalloc(&ft_wave_function_d,  npoints*sizeof(hipDoubleComplex));

            // Allocate space for device and managed scalars
            hipMalloc(&scattering_length_d,sizeof(double));
            hipMallocManaged(&norm_d,              sizeof(double));
            hipMallocManaged(&initial_norm_d,      sizeof(double));
            hipMallocManaged(&chemical_potential_d,sizeof(double));

            // Get the first necessary copies of input data from host to device
            hipMemcpy(external_potential_d,Vext.data(),       npoints*sizeof(double),         hipMemcpyHostToDevice);
            hipMemcpy(wave_function_d,     psi_0.data(),      npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d, &scattering_length,1      *sizeof(double),         hipMemcpyHostToDevice);

            // Initialize the mesh in Fourier space, and copy it to the device
            Vector<double> kx(nx);
            Vector<double> ky(ny);
            Vector<double> kz(nz);
            Vector<double> kmod2(nx,ny,nz);
            create_mesh_in_Fourier_space(x,y,z,kx,ky,kz);
            for (size_t i = 0; i < nx; ++i)
                for (size_t j = 0; j < ny; ++j)
                    for (size_t k = 0; k < nz; ++k)
                        kmod2(i,j,k) = std::pow(kx(i),2)+
                                       std::pow(ky(j),2)+
                                       std::pow(kz(k),2);
            hipMemcpy(kmod2_d, kmod2.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize space steps
            dx = x(1)-x(0);
            dy = y(1)-y(0);
            dz = z(1)-z(0);
            dv = dx*dy*dz;

            // Initialize the device reduce kernel
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();

            // Allocate temporary storage memory, required for reduction kernels
            hipMalloc(&temporary_storage_d,size_temporary_storage);
            hipDeviceSynchronize();

            // Calculate initial norm
            calculate_density(density_d,wave_function_d,npoints);
            hipDeviceSynchronize();
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
            hipDeviceSynchronize();
            norm_d[0]=norm_d[0]*dv;
            initial_norm_d[0]=norm_d[0];
            std::cout << "Initial norm: " << initial_norm_d[0] << std::endl;

            // Initialize the wave function to return as a result
            result_wave_function.reinit(nx,ny,nz);

            // Initialize the host vectors containing the mesh axis. This can be useful in particular for data output
            x_axis.reinit(nx);
            y_axis.reinit(ny);
            z_axis.reinit(nz);
            x_axis=x;
            y_axis=y;
            z_axis=z;
            kx_axis.reinit(nx);
            ky_axis.reinit(ny);
            kz_axis.reinit(nz);
            kx_axis=kx;
            ky_axis=ky;
            kz_axis=kz;
            hipMalloc(&x_axis_d,nx*sizeof(double));
            hipMalloc(&y_axis_d,ny*sizeof(double));
            hipMalloc(&z_axis_d,nz*sizeof(double));
            hipMemcpy(x_axis_d,x_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(y_axis_d,y_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(z_axis_d,z_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&kx_axis_d,nx*sizeof(double));
            hipMalloc(&ky_axis_d,ny*sizeof(double));
            hipMalloc(&kz_axis_d,nz*sizeof(double));
            hipMemcpy(kx_axis_d,kx_axis.data(),nx*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(ky_axis_d,ky_axis.data(),ny*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(kz_axis_d,kz_axis.data(),nz*sizeof(double),hipMemcpyHostToDevice);
            r2mod.reinit(nx,ny,nz);
            for(int i = 0; i < nx; ++i)
                for(int j = 0; j < ny; ++j)
                    for(int k = 0; k < nz; ++k)
                        r2mod(i,j,k) = std::pow(x(i),2);
            hipMalloc(&r2mod_d,npoints*sizeof(double));
            hipMemcpy(r2mod_d,r2mod.data(),npoints*sizeof(double),hipMemcpyHostToDevice);

            // Initialize the Fourier transform of the dipolar potential including a dipolar_cutoff
            // dipolar_cutoff contains cutoff_x, cutoff_y, cutoff_z
            // if all cutoffs are equal -> choose spherical cutoff
            // if two cutoffs are equal -> choose cylindrical cutoff
            // else return error
            hipMallocManaged(&epsilon_dd_d,sizeof(double));
            epsilon_dd_d[0] = 0.0;
            if(scattering_length != 0)
                epsilon_dd_d[0] = dipolar_length/scattering_length;
            Vtilde.reinit(nx,ny,nz);

            if(dipolar_cutoff[0] == dipolar_cutoff[1] && dipolar_cutoff[1] == dipolar_cutoff[2])
            {
                // prepare dipole potential with spherical cutoff
                 for (int i = 0; i < nx; ++i)
                 for (int j = 0; j < ny; ++j)
                 for (int k = 0; k < nz; ++k)
                    {
                       double aux = TWOPI * (
                                 kx[i]*sin(theta_mu)*cos(phi_mu) +
                                 ky[j]*sin(theta_mu)*sin(phi_mu)+
                                 kz[k]*cos(theta_mu));
                         double aux1 = TWOPI * sqrt(pow(kx[i], 2) + pow(ky[j], 2) + pow(kz[k], 2));
                         // double aux2 = sqrt(pow(kx[i], 2) + pow(ky[j], 2) + pow(kz[k], 2));
                         if (aux1 <= 1.E-6)
                            Vtilde(i,j,k) = 0;
                            //Vtilde(i,j,k) = -4*PI*scattering_length*epsilon_dd_d[0];
                             //Vtilde(i,j,k) = -4*PI*scattering_length*epsilon_dd_d[0]*(1+3*cos(dipolar_cutoff[0]*aux1)/(pow(dipolar_cutoff[0]*aux1,2))-3*sin(dipolar_cutoff[0]*aux1)/(pow(dipolar_cutoff[0]*aux1,3)));
                            //std::cout << Vtilde(i,j,k) << std::endl;
                         else
                            // Vtilde(i,j,k) = 12.0 * PI * scattering_length * epsilon_dd_d[0] * (pow(aux/aux1,2)-1.0/3.0);
                             Vtilde(i,j,k) = 12.0 * PI * scattering_length * epsilon_dd_d[0] * (pow(aux/aux1,2)-1.0/3.0)*(1+3*cos(dipolar_cutoff[0]*aux1)/(pow(dipolar_cutoff[0]*aux1,2))-3*sin(dipolar_cutoff[0]*aux1)/(pow(dipolar_cutoff[0]*aux1,3)));
                    }
                
            }
            else if(dipolar_cutoff[0] == dipolar_cutoff[1])
            {
                // prepare dipole potential with cylindrical cutoff (cylinder along the z-axis)
                
                int number_zeros = nx;
                double order = 0.0;
                int n_radial_int = 10;

                std::vector<double> zeros;
                // calculates zeroes of the J_0 bessel function
                boost::math::cyl_bessel_j_zero(order, 1, number_zeros, std::back_inserter(zeros));

                Vector<double> radial_vec(number_zeros);
                Vector<double> freq_vec(number_zeros);
                for (size_t i = 0; i < number_zeros; ++i)
                    {
                        radial_vec[i] = zeros[i]/dipolar_cutoff[0];
                    }

                double d_cyl_height = 2 * dipolar_cutoff[2]/ nz;
                double k_max = nz*TWOPI/(4*dipolar_cutoff[2]);
                double k_step = 2*k_max / (nz);

                Vector<double> kz_cutoff(nz);
                for (size_t i = 0; i < (nz); ++i) 
                    {
                    kz_cutoff[i] = -k_max + i * k_step;
                    }
                Vector<double> k_squared(number_zeros, nz);
                for (size_t i = 0; i < number_zeros; ++i)
                for (size_t j = 0; j < nz; ++j)
                        {
                        k_squared(i,j) = std::pow(radial_vec(i),2) +
                                         std::pow(kz_cutoff(j),2);
                        }

                Vector<double> cosine_sq(number_zeros, nz);
                Vector<double> sine_sq(number_zeros, nz);
                for (size_t i = 0; i < number_zeros; ++i)
                for (size_t j = 0; j < nz; ++j)
                    {
                    cosine_sq(i,j) = kz_cutoff(i)/k_squared(i,j);
                    sine_sq(i,j) = 1 - cosine_sq(i,j);
                    }

                // analytical cutoff for slice 0<z<Zmax, 0<r<Inf
                Vector<double> Vtilde_temp(number_zeros,nz);

                // change prefactor to 36.0 * PI * scattering_length *
                for (size_t i = 0; i < number_zeros; ++i)
                for (size_t j = 0; j < nz; ++j)
                    {
                        Vtilde_temp(i,j) = 36.0 * PI * scattering_length *(cosine_sq(i,j)-1/3
                                        +exp(-dipolar_cutoff[2]*kz_cutoff(j))   
                                        *(sine_sq(i,j)*cos(dipolar_cutoff[2]*kz_cutoff(j))    
                                        -sqrt(sine_sq(i,j)*cosine_sq(i,j))*sin(dipolar_cutoff[2]*kz_cutoff(j))));
                    }
                // integration over 0<z<Zmax, Rmax<r<R_lim*Rmax
                int R_lim = 2000;
                double dr =(R_lim-1)*dipolar_cutoff[0]/n_radial_int;
                Vector<double> radial_integration_coordinate(n_radial_int);
                for (size_t i = 0; i < n_radial_int; ++i)
                    {
                        radial_integration_coordinate(i) = (static_cast<double>(i + 1) - 0.5) * dr + dipolar_cutoff[0];
                    }
                Vector<double> height_integration_coordinate(nz);
                for (size_t i = 0; i < nz; ++i)
                    {
                        height_integration_coordinate(i) = (static_cast<double>(i + 1) - 0.5) * dz/2 ;
                    }

                Vector<double> r_squared(n_radial_int, nz);
                for (size_t i = 0; i < n_radial_int; ++i)
                for (size_t j = 0; j < nz; ++j)
                        {
                        r_squared(i,j) = std::pow(radial_integration_coordinate(i),2) +
                                         std::pow(height_integration_coordinate(j),2);
                        }
                
                Vector<double> interaction_real(n_radial_int, nz);

                for (size_t i = 0; i < n_radial_int; ++i)
                for (size_t j = 0; j < nz; ++j)
                    {
                        interaction_real(i,j) = (1-3*std::pow(height_integration_coordinate(j),2)/r_squared(i,j))/std::pow(r_squared(i,j), 3/2);
                    }


                Vector<double> besselr_interm (number_zeros*n_radial_int);
                for (size_t j = 0; j < n_radial_int; ++j)
                for (size_t i = 0; i < number_zeros; ++i)
                    {
                        besselr_interm(i*n_radial_int+j) = radial_integration_coordinate[j]*boost::math::cyl_bessel_j(0,radial_vec[i]*radial_integration_coordinate[j]) ;
                    }
                
                Vector<double> besselr (number_zeros*n_radial_int, nz);
                for (size_t i = 0; i < number_zeros*n_radial_int; ++i){
                     // (for (size_t j = 0; j < number_zeros*n_radial_int; ++j){
                    for (size_t k = 0; k < nz; ++k)
                    {
                        besselr(i, k) = besselr_interm(i)  ;
                    }}
                
                Vector<double> igbz(nz, nz);
                for (size_t i = 0; i < (nz); ++i) 
                for (size_t j = 0; j < (nz); ++j) 
                    {
                    igbz(i,j) = cos(kz_cutoff(i)*height_integration_coordinate(j)) ;
                    }


                /*
                for (size_t i = 0; i < number_zeros; ++i)
                    {
                        std::cout <<"Rad vec"<<radial_vec[i] << std::endl ;
                    } 


                 for (size_t i = 0; i < number_zeros; ++i)
                    {
                        std::cout << "Rad int coord"<< radial_integration_coordinate[i] << std::endl ;
                    } 

                for (size_t i = 0; i < number_zeros; ++i)
                    {
                        std::cout << "Rad int * Rad vec"<< radial_vec(i)*radial_integration_coordinate(i) << std::endl ;
                    } 
                /**/    
                for (size_t i = 0; i < (nz); ++i) 
                    {
                    std::cout << height_integration_coordinate[i] << std::endl;
                    }
                /* 
                for (size_t i = 0; i < nz; ++i)
                for (size_t j = 0; j < nz*number_zeros; ++j)
                    {
                    {
                        integrand =   
                        for (size_t k = 0; i < number_zeros; ++i)
                            {


                        
                            } 

                    }
                    } 
                    */

            }
            else
            {
                std::cout << "No dipolar potential is implemented for this choice of cutoffs" << std::endl;
            }
            
            
            
            // for (int i = 0; i < nx; ++i)
            //     for (int j = 0; j < ny; ++j)
            //         for (int k = 0; k < nz; ++k)
            //         {
            //             double aux = TWOPI * (
            //                     kx[i]*sin(theta_mu)*cos(phi_mu) +
            //                     ky[j]*sin(theta_mu)*sin(phi_mu)+
            //                     kz[k]*cos(theta_mu));
            //             double aux1 = TWOPI * sqrt(pow(kx[i], 2) + pow(ky[j], 2) + pow(kz[k], 2));
            //             if (aux1 <= 1.E-6)
            //                 Vtilde(i,j,k) = -4*PI*scattering_length*epsilon_dd_d[0];
            //             else
            //                 Vtilde(i,j,k) =
            //                         12.0 * PI * scattering_length * epsilon_dd_d[0] * (pow(aux/aux1,2)-1.0/3.0);
            //         }
            

            
            hipMalloc(&Vtilde_d,npoints*sizeof(hipDoubleComplex));
            hipMemcpy(Vtilde_d,Vtilde.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMalloc(&Phi_dd_d,npoints*sizeof(hipDoubleComplex));

            // Initialize gamma(\epsilon_dd) for the LHY correction
            hipMallocManaged(&gamma_epsilondd_d,sizeof(double));
            gamma_epsilondd_d[0] = 0.0;
            if (epsilon_dd_d[0] != 0 && add_lhy_correction)
            {
                gamma_epsilondd_d[0] = 64.0*sqrt(PI)/3*sqrt(pow(scattering_length,5));
                double F_epsilon_dd=0.0;
                int n_theta=1000;
                double d_theta=PI/(n_theta-1);

                std::complex<double> csum;
                std::complex<double> caux;
                csum={0.0,0.0};
                for (int i = 0; i < n_theta; ++i)
                {
                    double theta=i*d_theta;
                    caux = pow(1.0+epsilon_dd_d[0]*(3.0*pow(cos(theta),2)-1.0),5);
                    caux = sqrt(caux);
                    csum += sin(theta)*caux;
                }
                csum *= d_theta;
                F_epsilon_dd = csum.real();
                gamma_epsilondd_d[0] *= F_epsilon_dd;
            }

            // Initialize the wave function for the output
            wave_function_output.reinit(nx,ny,nz);

        }

        /**
         * @brief Destructor frees device memory
         *
         * */

        DipolarGPSolver::~DipolarGPSolver()
        {
            hipFree(external_potential_d);
            hipFree(density_d);
            hipFree(norm_d);
            hipFree(initial_norm_d);
            hipFree(wave_function_d);
            hipFree(ft_wave_function_d);
            hipFree(hpsi_d);
            hipFree(x_axis_d);
            hipFree(y_axis_d);
            hipFree(z_axis_d);
            hipFree(kx_axis_d);
            hipFree(ky_axis_d);
            hipFree(kz_axis_d);
            hipFree(kmod2_d);
            hipFree(r2mod_d);
            hipFree(chemical_potential_d);
            hipFree(scattering_length_d);
            hipFree(alpha_d);
            hipFree(beta_d);
            hipFree(time_step_d);
            hipFree(temporary_storage_d);
            hipFree(Vtilde_d);
            hipFree(Phi_dd_d);
            hipFree(epsilon_dd_d);
            hipFree(gamma_epsilondd_d);
        }

        /**
         * @brief Get a pointer to the wave function stored in the device
         * */

        hipDoubleComplex* DipolarGPSolver::get_wave_function_device_pointer()
        {
            return wave_function_d;
        }

        /**
         * @brief Get a pointer to the wave function stored on the device
         * */

        hipDoubleComplex* DipolarGPSolver::get_ft_wave_function_device_pointer()
        {
            return ft_wave_function_d;
        }

        /**
         * @brief Calculate the density profile
         *
         * */

        void DipolarGPSolver::calculate_density(double *density, hipDoubleComplex *wave_function,int size)
        {
            SimpleKernels::square_vector<<<gridSize,blockSize>>>(density,wave_function,size);
        }

        /**
         *
         * @brief Run the gradient descent
         *
         * \warning No check of the residual!
         *
         * */

        std::tuple<Vector<std::complex<double>>, double>
        DipolarGPSolver::run_gradient_descent(int max_num_iter,
                                              double alpha,
                                              double beta,
                                              std::ostream &output_stream,
                                              int write_output_every)

        {
            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);

            //--------------------------------------------------//
            //    Here the gradient-descent iterations start    //
            //--------------------------------------------------//

            // Allocate space for some new data on the device
            hipMalloc(&alpha_d,sizeof(double));
            hipMemcpy(alpha_d,&alpha,sizeof(double),hipMemcpyHostToDevice);
            hipMalloc(&beta_d,sizeof(double));
            hipMemcpy(beta_d,&beta,sizeof(double),hipMemcpyHostToDevice);
            hipDoubleComplex* psi_new;
            hipDoubleComplex* psi_old;
            hipMalloc(&psi_new,npoints*sizeof(hipDoubleComplex));
            hipMalloc(&psi_old,npoints*sizeof(hipDoubleComplex));
            hipDoubleComplex* c_density_d;
            hipMalloc(&c_density_d,npoints*sizeof(hipDoubleComplex));

            // Loop starts here
            for (int it = 0; it < max_num_iter; ++it)
            {

                // Calculate the action of the laplacian
                hipfftExecZ2Z(ft_plan, wave_function_d, ft_wave_function_d, HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,kmod2_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan, ft_wave_function_d, hpsi_d, HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(hpsi_d,0.5*pow(TWOPI,2)/npoints,npoints);
                hipDeviceSynchronize();

                // Calculate the dipolar potential
                SimpleKernels::square_vector<<<gridSize,blockSize>>>(c_density_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,c_density_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                               Vtilde_d,
                                                                               npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,Phi_dd_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(Phi_dd_d,1./npoints,npoints);

                // Calculate the rest of H|psi>
                SolverKernels::step_2_dipolar_hpsi<<<gridSize,blockSize>>>(hpsi_d,
                                                                             wave_function_d,
                                                                             external_potential_d,
                                                                             Phi_dd_d,
                                                                             scattering_length_d,
                                                                             gamma_epsilondd_d,
                                                                             npoints);
                hipDeviceSynchronize();

                // Perform a gradient descent (plus heavy-ball) step
                SolverKernels::gradient_descent_step<<<gridSize,blockSize>>>(wave_function_d,
                                                                               hpsi_d,
                                                                               psi_new,
                                                                               psi_old,
                                                                               alpha_d,
                                                                               beta_d,
                                                                               npoints);
                hipDeviceSynchronize();

                // Normalize the wave function
                SimpleKernels::square_vector<<<gridSize,blockSize>>>(density_d,psi_new,npoints);
                hipDeviceSynchronize();
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,norm_d,npoints);
                hipDeviceSynchronize();
                norm_d[0] = norm_d[0]*dv;
                SimpleKernels::rescale<<<gridSize,blockSize>>>(wave_function_d,
                                                                 psi_new,
                                                                 sqrt(initial_norm_d[0]/norm_d[0]),
                                                                 npoints);
                hipDeviceSynchronize();

                // Calculate the chemical potential
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(density_d,hpsi_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,density_d,chemical_potential_d,npoints);
                hipDeviceSynchronize();
                chemical_potential_d[0] = chemical_potential_d[0]*dv/norm_d[0];

                // Eventually write some output
                if(it % write_output_every == 0)
                    write_gradient_descent_output(it,output_stream);

            }

            // Free the remaining arrays from the device
            hipFree(psi_new);
            hipFree(psi_old);
            hipFree(c_density_d);

            // Copy out the results
            hipMemcpy(result_wave_function.data(),
                       wave_function_d,
                       npoints*sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost);
            double result_chemical_potential = chemical_potential_d[0];

            // Return
            return std::make_pair(result_wave_function,result_chemical_potential);

        }

        /**
         *
         * @brief Write gradient descent output
         *
         * */

        void DipolarGPSolver::write_gradient_descent_output(size_t        iteration_number,
                                                            std::ostream& output_stream)
        {
            output_stream << iteration_number << " " << chemical_potential_d[0] << std::endl;
        }

        /**
         * @brief Real-time operator splitting
         */

        void DipolarGPSolver::run_operator_splitting(int number_of_time_steps,
                                                     double time_step,
                                                     std::ostream &output_stream,
                                                     int write_output_every,
						     int iteration_twa)
        {
            // Copy input data into the device
            hipMallocManaged(&time_step_d,sizeof(double));
            hipMemcpy(time_step_d,&time_step,sizeof(double),hipMemcpyHostToDevice);

            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);
            hipDoubleComplex* c_density_d;
            hipMalloc(&c_density_d,npoints*sizeof(hipDoubleComplex));

            // Initialize other variables
            this->write_output_every=write_output_every;
	        this->iteration_twa=iteration_twa;

            //----------------------------------------------------//
            //    Here the operator-splitting iterations start    //
            //----------------------------------------------------//
            for (size_t it = 0; it < number_of_time_steps; ++it)
            {

                // Write output starting from the very first iteration
                if(it % write_output_every == 0)
                    write_operator_splitting_output(it,output_stream);

                // Calculate the current value of dipolar potential
                SimpleKernels::square_vector<<<gridSize,blockSize>>>(c_density_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,c_density_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                               Vtilde_d,
                                                                               npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,Phi_dd_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(Phi_dd_d,1./npoints,npoints);
                hipDeviceSynchronize();

                // Solve step-1 of operator splitting, i.e. the one NOT involving Fourier transforms
                SolverKernels::step_1_operator_splitting_dipolars<<<gridSize,blockSize>>>(wave_function_d,
                                                                                            external_potential_d,
                                                                                            Phi_dd_d,
                                                                                            time_step_d,
                                                                                            scattering_length_d,
                                                                                            gamma_epsilondd_d,
                                                                                            npoints);
                hipDeviceSynchronize();

                // Solve step-2 of operator splitting, i.e. the one actually involving Fourier transforms
                hipfftExecZ2Z(ft_plan,wave_function_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SolverKernels::aux_step_2_operator_splitting<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                                       kmod2_d,
                                                                                       time_step_d,
                                                                                       npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,wave_function_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(wave_function_d,1./npoints,npoints);
                hipDeviceSynchronize();

            }
            hipFree(c_density_d);
        }

        /**
         * @brief Overloaded Real-time operator splitting allowing to start at index_start
         */

        void DipolarGPSolver::run_operator_splitting(int number_of_time_steps,
                                                     int index_start,
                                                     double time_step,
                                                     std::ostream &output_stream,
                                                     int write_output_every,
						                             int iteration_twa)
        {
            // Copy input data into the device
            hipMallocManaged(&time_step_d,sizeof(double));
            hipMemcpy(time_step_d,&time_step,sizeof(double),hipMemcpyHostToDevice);

            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);
            hipDoubleComplex* c_density_d;
            hipMalloc(&c_density_d,npoints*sizeof(hipDoubleComplex));

            // Initialize other variables
            this->write_output_every=write_output_every;
	        this->iteration_twa=iteration_twa;

            //----------------------------------------------------//
            //    Here the operator-splitting iterations start    //
            //----------------------------------------------------//
            for (size_t it = index_start; it < number_of_time_steps + index_start; ++it)
            {

                // Write output starting from the very first iteration
                if(it % write_output_every == 0)
                    write_operator_splitting_output(it,output_stream);

                // Calculate the current value of dipolar potential
                SimpleKernels::square_vector<<<gridSize,blockSize>>>(c_density_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,c_density_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                               Vtilde_d,
                                                                               npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,Phi_dd_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(Phi_dd_d,1./npoints,npoints);
                hipDeviceSynchronize();

                // Solve step-1 of operator splitting, i.e. the one NOT involving Fourier transforms
                SolverKernels::step_1_operator_splitting_dipolars<<<gridSize,blockSize>>>(wave_function_d,
                                                                                            external_potential_d,
                                                                                            Phi_dd_d,
                                                                                            time_step_d,
                                                                                            scattering_length_d,
                                                                                            gamma_epsilondd_d,
                                                                                            npoints);
                hipDeviceSynchronize();

                // Solve step-2 of operator splitting, i.e. the one actually involving Fourier transforms
                hipfftExecZ2Z(ft_plan,wave_function_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SolverKernels::aux_step_2_operator_splitting<<<gridSize,blockSize>>>(ft_wave_function_d,
                                                                                       kmod2_d,
                                                                                       time_step_d,
                                                                                       npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,wave_function_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(wave_function_d,1./npoints,npoints);
                hipDeviceSynchronize();

            }
            hipFree(c_density_d);
        }

        /**
         * @brief Real-time operator splitting for Faraday patterns
         */

        void DipolarGPSolver::run_operator_splitting_faraday(int number_of_time_steps,
                                                             double time_step,
                                                             double modulation_amplitude,
                                                             double modulation_frequency,
                                                             std::ostream &output_stream,
                                                             int write_output_every)
        {

            // Copy input data into the device
            hipMallocManaged(&time_step_d,sizeof(double));
            hipMemcpy(time_step_d,&time_step,sizeof(double),hipMemcpyHostToDevice);

            // Initialize the fft plan required for the calculation of the laplacian
            hipfftHandle ft_plan;
            if(problem_is_2d)
                hipfftPlan2d(&ft_plan,nx,ny,HIPFFT_Z2Z);
            else if(problem_is_3d)
                hipfftPlan3d(&ft_plan,nx,ny,nz,HIPFFT_Z2Z);
            hipDoubleComplex* c_density_d;
            hipMalloc(&c_density_d,npoints*sizeof(hipDoubleComplex));

            // Initialize other variables
            this->write_output_every=write_output_every;
            double initial_scattering_length;
            double current_scattering_length;
            hipMemcpy(&initial_scattering_length,scattering_length_d,sizeof(double),hipMemcpyDeviceToHost);

            //----------------------------------------------------//
            //    Here the operator-splitting iterations start    //
            //----------------------------------------------------//
            for (size_t it = 0; it < number_of_time_steps; ++it)
            {

                // Write output starting from the very first iteration
                if(it % write_output_every == 0)
                    write_operator_splitting_output(it,output_stream);

                // Calculate the current value of dipolar potential
                SimpleKernels::square_vector<<<gridSize,blockSize>>>(c_density_d,wave_function_d,npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,c_density_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SimpleKernels::vector_multiplication<<<gridSize,blockSize>>>(ft_wave_function_d,
                        Vtilde_d,
                        npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,Phi_dd_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(Phi_dd_d,1./npoints,npoints);
                hipDeviceSynchronize();

                // Solve step-1 of operator splitting, i.e. the one NOT involving Fourier transforms
                double current_time = it*time_step;
                current_scattering_length =
                        initial_scattering_length*(1.0+modulation_amplitude*std::cos(TWOPI*modulation_frequency*current_time));
                hipMemcpy(scattering_length_d,&current_scattering_length,sizeof(double),hipMemcpyHostToDevice);

                SolverKernels::step_1_operator_splitting_dipolars<<<gridSize,blockSize>>>(wave_function_d,
                        external_potential_d,
                        Phi_dd_d,
                        time_step_d,
                        scattering_length_d,
                        gamma_epsilondd_d,
                        npoints);
                hipDeviceSynchronize();

                // Solve step-2 of operator splitting, i.e. the one actually involving Fourier transforms
                hipfftExecZ2Z(ft_plan,wave_function_d,ft_wave_function_d,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                SolverKernels::aux_step_2_operator_splitting<<<gridSize,blockSize>>>(ft_wave_function_d,
                        kmod2_d,
                        time_step_d,
                        npoints);
                hipDeviceSynchronize();
                hipfftExecZ2Z(ft_plan,ft_wave_function_d,wave_function_d,HIPFFT_BACKWARD);
                hipDeviceSynchronize();
                SimpleKernels::rescale<<<gridSize,blockSize>>>(wave_function_d,1./npoints,npoints);
                hipDeviceSynchronize();

            }
            hipFree(c_density_d);
        }

        /**
         * @brief Operator splitting output.
         *
         * This function is called after a copy of the current wave function outside of the GPU, is such a way that it
         * can be used for example for data analysis or to write it to a file for visualization. Since each call
         * blocks the real-time evolution on the GPU until the function has finished, it is better to use it with
         * moderation to avoid a big loss of performance.
         *
         * */

        void DipolarGPSolver::write_operator_splitting_output(size_t        iteration_number,
                                                              std::ostream& output_stream)
        {}

        /**
         * @brief Copy the wave function out from device to host
         *
         * In derived classes, the wave function will be available as wave_function_output
         *
         * */

        void DipolarGPSolver::copy_out_wave_function()
        {
            hipMemcpy(wave_function_output.data(),
                       wave_function_d,
                       npoints*sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        }

        /**
         * @brief Reinitialize the solver with new external potential and wave function
         *
         * */

        void DipolarGPSolver::reinit(Vector<double> &Vext,Vector<std::complex<double>> &psi)
        {
            hipMemcpy(wave_function_d,psi.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(external_potential_d,Vext.data(),npoints*sizeof(double),hipMemcpyHostToDevice);
        }

        /**
         *
         * @brief Reinitialize the solver with new external potential, wave function and scattering length.
         *
         * */

        void DipolarGPSolver::reinit(Vector<double> &Vext,Vector<std::complex<double>> &psi,double scattering_length)
        {
            hipMemcpy(wave_function_d,psi.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
            hipMemcpy(external_potential_d,Vext.data(),npoints*sizeof(double),hipMemcpyHostToDevice);
            hipMemcpy(scattering_length_d,&scattering_length,sizeof(double),hipMemcpyHostToDevice);
        }

        /**
         * @brief Set initial conditions for a Truncated Wigner run
         *
         */

        void DipolarGPSolver::set_tw_initial_conditions(bool system_is_trapped,
                                                        std::default_random_engine& generator)
        {

            // Need to get in a copy of the scattering length
            double scattering_length;
            hipMemcpy(&scattering_length,scattering_length_d,sizeof(double),hipMemcpyDeviceToHost);

            // First, consider the case in which the system is not trapped

            if (!system_is_trapped)
            {

                // Obtain a random seed from the clock
                // std::default_random_engine generator;
                // typedef std::chrono::high_resolution_clock clock;
                // clock::time_point beginning = clock::now();
                // clock::duration d = clock::now() - beginning;
                // generator.seed(d.count());
                std::uniform_real_distribution<double> distribution(-1, 1);

                // Generate the alphas
                double u, v, s; // useful additional variables that we use to generate our random numbers
                std::complex<double> alphak;

                // Now refill the initial wave function with the single particle modes
                if (problem_is_2d)
                {

                    Vector<std::complex<double>> psitilde_tw(nx, ny);
                    Vector<std::complex<double>> psi(nx, ny);
                    MKLWrappers::DFtCalculator dft_tw(psi, psitilde_tw);

                    hipMemcpy(psi.data(), wave_function_d, npoints * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

                    dft_tw.compute_forward();

                    double density = initial_norm_d[0] / (4. * x_axis(nx - 1) * y_axis(ny - 1));
                    double eps_k;
                    std::complex<double> Ek, uk, vk;

                    for (int i = 0; i < nx; ++i)
                        for (int j = 0; j < ny; ++j)
                        {
                            do
                            {
                                u = distribution(generator);
                                v = distribution(generator);
                                s = u * u + v * v;
                            } while (s >= 1.0 || s == 0);

                            s = sqrt((-2.0 * log(s)) / s);
                            u = u * s;
                            v = v * s;
                            alphak.real(std::sqrt(0.25) * u);
                            alphak.imag(std::sqrt(0.25) * v);

                            // eps_k = 0.5*std::sqrt(pow(kx[i],2)+pow(ky[j],2));
                            // Ek = std::sqrt(eps_k*(eps_k+2*density*(4*PI*scattering_length+Vtilde(i,j))));

                            // changed single particle energy to k^2/m, and added the 2pi factor
                            // 4pi -> sqrt(8pi) since we are in quasi 2d

                            eps_k = 0.5*(pow(TWOPI*kx_axis[i],2)+pow(TWOPI*ky_axis[j],2));
                            Ek = std::sqrt(eps_k*(eps_k+2*density*(std::sqrt(4*TWOPI)*scattering_length + Vtilde(i,j))));

                            if (eps_k == 0)
                            {
                                uk = 1;
                                vk = 0;
                            }
                            else
                            {
                                uk = 0.5 * (sqrt(eps_k / Ek) + sqrt(Ek / eps_k));
                                vk = 0.5 * (sqrt(eps_k / Ek) - sqrt(Ek / eps_k));
                            }

                            psitilde_tw(i, j) += (alphak * uk - conj(alphak) * vk);

                        }

                    dft_tw.compute_backward();

                    hipMemcpy(wave_function_d,psi.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);

                }
                else if (problem_is_3d)
                {

                    Vector<std::complex<double>> psitilde_tw(nx,ny,nz);
                    Vector<std::complex<double>> psi(nx,ny,nz);
                    MKLWrappers::DFtCalculator dft_tw(psi,psitilde_tw);

                    hipMemcpy(psi.data(), wave_function_d, npoints * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

                    dft_tw.compute_forward();

                    double density = initial_norm_d[0] / (4. * x_axis(nx - 1) * y_axis(ny - 1) * z_axis(nz-1));
                    double eps_k;
                    std::complex<double> Ek, uk, vk;

                    for (int i = 0; i < nx; ++i)
                        for (int j = 0; j < ny; ++j)
                            for(int k = 0; k < nz; ++k)
                            {
                                do
                                {
                                    u = distribution(generator);
                                    v = distribution(generator);
                                    s = u * u + v * v;
                                } while (s >= 1.0 || s == 0);

                                s = sqrt((-2.0 * log(s)) / s);
                                u = u * s;
                                v = v * s;
                                alphak.real(std::sqrt(0.25) * u);
                                alphak.imag(std::sqrt(0.25) * v);

                                eps_k = 0.5 * std::sqrt(pow(kx_axis[i], 2) + pow(ky_axis[j], 2) + pow(kz_axis[k],2));
                                Ek = std::sqrt(eps_k*(eps_k+2*density*(4*PI*scattering_length+Vtilde(i, j, k))));
                                if (eps_k == 0)
                                {
                                    uk = 1;
                                    vk = 0;
                                }
                                else
                                {
                                    uk = 0.5 * (sqrt(eps_k / Ek) + sqrt(Ek / eps_k));
                                    vk = 0.5 * (sqrt(eps_k / Ek) - sqrt(Ek / eps_k));
                                }

                                psitilde_tw(i,j,k) += (alphak * uk - conj(alphak) * vk);

                            }

                    dft_tw.compute_backward();

                    hipMemcpy(wave_function_d,psi.data(),npoints*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);

                }

                // We are now ready to perform a new run of the TWA.
            }
        }

        /**
          * @brief Evaluate integrated momentum distributions for a three-dimensional problem.
          *
          * */

        void DipolarGPSolver::evaluate_integrated_occupation_number()
        {
            // If this is the first time that the function is called, it needs to set up accordingly some internal
            // control variable, plus it needs to initialize some fata
            if(first_call_evaluate_integrated_occupation_number)
            {
                first_call_evaluate_integrated_occupation_number = false;
                hipfftPlan3d(&dft_handle_integrated_occupation_number,nx,ny,nz,HIPFFT_Z2Z);
                integrated_occupation_number.resize(3);
                integrated_occupation_number[0].reinit(nx);
                integrated_occupation_number[1].reinit(ny);
                integrated_occupation_number[2].reinit(nz);
                hipMalloc(&xlayer_d,ny*nz*sizeof(double));
                hipMalloc(&ylayer_d,nx*nz*sizeof(double));
                hipMalloc(&zlayer_d,nx*ny*sizeof(double));
                hipMallocManaged(&x_integral,sizeof(double));
                hipMallocManaged(&y_integral,sizeof(double));
                hipMallocManaged(&z_integral,sizeof(double));
                hipMalloc(&occupation_number_d,nx*ny*nz*sizeof(double));
            }

            // Execute the FFT
            hipfftExecZ2Z(dft_handle_integrated_occupation_number,
                         wave_function_d,
                         ft_wave_function_d,
                         HIPFFT_FORWARD);
            hipDeviceSynchronize();

            calculate_density(occupation_number_d,ft_wave_function_d,npoints);
            hipDeviceSynchronize();

            // Calculate integrated occupation number along x, i.e. \int dydz |\psi_k(x,y,z)|^2
            dim3 threadsPerBlock_x(16,16);
            dim3 numBlocks_x((ny + threadsPerBlock_x.x -1) / threadsPerBlock_x.x,
                             (nz + threadsPerBlock_x.y -1) / threadsPerBlock_x.y);
            for(int layer_number = 0; layer_number < nx; ++layer_number)
            {
                SimpleKernels::extract_layer_x<<<threadsPerBlock_x,numBlocks_x>>>(xlayer_d,
                                                                                  occupation_number_d,
                                                                                  layer_number,
                                                                                  nx,ny,nz);
                hipDeviceSynchronize();

                x_integral[0] = 0.0;
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,xlayer_d,x_integral,ny*nz);
                hipDeviceSynchronize();
                integrated_occupation_number[0][layer_number] = x_integral[0];
            }

            // Calculate integrated occupation number along y, i.e. \int dxdz |\psi_k(x,y,z)|^2
            dim3 threadsPerBlock_y(16,16);
            dim3 numBlocks_y((nx + threadsPerBlock_y.x -1) / threadsPerBlock_y.x,
                             (nz + threadsPerBlock_y.y -1) / threadsPerBlock_y.y);
            for(int layer_number = 0; layer_number < ny; ++layer_number)
            {
                SimpleKernels::extract_layer_y<<<threadsPerBlock_y,numBlocks_y>>>(ylayer_d,
                                                                                  occupation_number_d,
                                                                                  layer_number,
                                                                                  nx,ny,nz);
                hipDeviceSynchronize();

                y_integral[0] = 0.0;
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,ylayer_d,y_integral,nx*nz);
                hipDeviceSynchronize();
                integrated_occupation_number[1][layer_number] = y_integral[0];
            }

            // Calculate integrated occupation number along z, i.e. \int dxdy |\psi_k(x,y,z)|^2
            dim3 threadsPerBlock_z(16,16);
            dim3 numBlocks_z((nx + threadsPerBlock_z.x -1) / threadsPerBlock_z.x,
                             (ny + threadsPerBlock_z.y -1) / threadsPerBlock_z.y);
            for(int layer_number = 0; layer_number < nz; ++layer_number)
            {
                SimpleKernels::extract_layer_z<<<threadsPerBlock_z,numBlocks_z>>>(zlayer_d,
                                                                                  occupation_number_d,
                                                                                  layer_number,
                                                                                  nx,ny,nz);
                hipDeviceSynchronize();

                z_integral[0] = 0.0;
                hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,zlayer_d,z_integral,nx*ny);
                hipDeviceSynchronize();
                integrated_occupation_number[2][layer_number] = z_integral[0];
            }

        }

        /**
         * @brief Apply a low-pass filter on the wave function
         *
         * This function is useful for example for visualizing vortex tangles
         *
         * */

        void DipolarGPSolver::apply_momentum_cutoff(double kc)
        {

            if(first_call_low_pass_filter)
            {
                first_call_low_pass_filter = false;
                hipfftPlan3d(&dft_quasi_condensate,nx,ny,nz,HIPFFT_Z2Z);
                hipMalloc(&wave_function_quasi_condensate_d,npoints*sizeof(hipDoubleComplex));
                hipMalloc(&ft_wave_function_quasi_condensate_d,npoints*sizeof(hipDoubleComplex));
                wave_function_quasi_condensate.reinit(nx,ny,nz);
            }

            SimpleKernels::low_pass_filter<<<gridSize,blockSize>>>(ft_wave_function_quasi_condensate_d,
                                                                   ft_wave_function_d,
                                                                   kmod2_d,
                                                                   kc,
                                                                   npoints);
            hipDeviceSynchronize();
            hipfftExecZ2Z(dft_quasi_condensate,
                         ft_wave_function_quasi_condensate_d,
                         wave_function_quasi_condensate_d,
                         HIPFFT_BACKWARD);
            hipDeviceSynchronize();

            SimpleKernels::rescale<<<gridSize,blockSize>>>(wave_function_quasi_condensate_d,1./npoints,npoints);

            hipMemcpy(wave_function_quasi_condensate.data(),
                       wave_function_quasi_condensate_d,
                       npoints*sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

        }

        /**
         * @brief Apply a density threshold on a wave function.
         *
         * This is useful to eventually isolate a vortex tangle.
         *
         * */

        void DipolarGPSolver::calculate_vortex_tangle_length(double n_threshold)
        {
            if(first_call_calculate_vortex_tangle_length)
            {
                first_call_calculate_vortex_tangle_length=false;
                hipMalloc(&vortex_tangle_density_d,npoints*sizeof(double));
                vortex_tangle_density.reinit(nx,ny,nz);
                hipMallocManaged(&vortex_tangle_length,sizeof(double));
                file_vortex_tangle_length.open("vortex_tangle_length.txt",std::ios_base::app);
            }

            // Apply density threshold
            SimpleKernels::density_threshold<<<gridSize,blockSize>>>(wave_function_quasi_condensate_d,
                                                                     vortex_tangle_density_d,
                                                                     n_threshold,
                                                                     npoints);
            hipDeviceSynchronize();

            // Eventually copy out the full vortex tangle
            hipMemcpy(vortex_tangle_density.data(),
                       vortex_tangle_density_d,
                       npoints*sizeof(double),
                       hipMemcpyDeviceToHost);

            // Calculate the total length of the tangle
            vortex_tangle_length[0] = 0.0;
            hipcub::DeviceReduce::Sum(temporary_storage_d,size_temporary_storage,vortex_tangle_density_d,
                                   vortex_tangle_length,ny*nz*nz);
            hipDeviceSynchronize();
            file_vortex_tangle_length << vortex_tangle_length_time_iteration << " "
                                      << vortex_tangle_length[0]/npoints << std::endl;
            vortex_tangle_length_time_iteration += 1;
        }
    }
}
