#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
*
*    This file is part of the UltraCold project.
*
*    UltraCold is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    any later version.
*    UltraCold is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*    You should have received a copy of the GNU General Public License
*    along with UltraCold.  If not, see <https://www.gnu.org/licenses/>.
*
*--------------------------------------------------------------------------------*/

#include "simple_kernels.cuh"
#define PI 3.1415926535897932384626433
#define TWOPI (2*PI)

namespace UltraCold
{
    namespace SimpleKernels
    {

        ///////////////////////////////////////////////////////
        // Pure device kernels
        ///////////////////////////////////////////////////////

        /**
         * @brief A useful complex exponential function
         **/

        __device__ hipDoubleComplex complex_exponential(hipDoubleComplex input)
        {
            hipDoubleComplex res;
            double t = expf (input.x);
            sincos (input.y, &res.y, &res.x);
            res.x *= t;
            res.y *= t;
            return res;
        }

        ////////////////////////////////////////////////////
        // Global kernels
        ////////////////////////////////////////////////////

        /**
         *
         * @brief Calculate the square of a complex vector, storing the result in another complex vector
         *
         * */

        __global__ void square_vector(hipDoubleComplex* result,
                                      hipDoubleComplex* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = input[i].x*input[i].x +
                              input[i].y*input[i].y;
                result[i].y = 0.0;
            }
        }
        /**
         *
         * @brief Calculate the square of a complex vector
         *
         * */

        __global__ void square_vector(double* result,
                                      hipDoubleComplex* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input[i].x*input[i].x +
                            input[i].y*input[i].y;
            }
        }

        /**
         *
         * @brief Calculate the square of a real vector
         *
         * */

        __global__ void square_vector(double* result,
                                      double* input,
                                      int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input[i]*input[i];
            }
        }

        /**
         *
         * @brief Multiply a real vector times the square of a complex vector
         *
         * */

        __global__ void vector_average(double* result,
                                       double* input1,
                                       hipDoubleComplex* input2,
                                       int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input1[i]*(input2[i].x*input2[i].x+input2[i].y*input2[i].y);
            }
        }

        /**
         *
         * @brief Multiply two complex vectors. Overwrite the first one
         *
         * */

        __global__ void vector_addition(hipDoubleComplex* result,
                                        hipDoubleComplex* input,
                                        int size)
        {
            //hipDoubleComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                //temp = result[i];
                result[i].x = result[i].x + input[i].x;
                result[i].y = result[i].y + input[i].y;
            }
        }

        /**
         *
         * @brief Multiply two complex vectors. Overwrite the first one
         *
         * */

        __global__ void vector_multiplication(hipDoubleComplex* result,
                                              hipDoubleComplex* input,
                                              int size)
        {
            hipDoubleComplex temp;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                temp = result[i];
                result[i].x = temp.x*input[i].x - temp.y*input[i].y;
                result[i].y = temp.y*input[i].x + temp.x*input[i].y;
            }
        }

        /**
         *
         * @brief Multiply a complex and a real vector. Overwrite the complex one
         *
         * */

        __global__ void vector_multiplication(hipDoubleComplex* result,
                                              double* input,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = result[i].x*input[i];
                result[i].y = result[i].y*input[i];
            }
        }

        /**
          *
          * @brief Multiply two real vectors. Overwrite the first one
          *
          * */

        __global__ void vector_multiplication(double* result,
                                              double* input,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = result[i]*input[i];
            }
        }

        /**
         *
         * @brief Multiply two complex vectors in the case in which the result is a real one
         *
         * */

        __global__ void vector_multiplication(double* result,
                                              hipDoubleComplex* input1,
                                              hipDoubleComplex* input2,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i] = input1[i].x * input2[i].x;
            }
        }

        /**
         *
         * @brief Rescale a vector for a given input scalar
         *
         * */

        __global__ void rescale(hipDoubleComplex* result,
                                double input,
                                int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = result[i].x * input;
                result[i].y = result[i].y * input;
            }

        }

        /**
         *
         * @brief Rescale a vector for a given input scalar, storing the result in another vector
         *
         * */

        __global__ void rescale(hipDoubleComplex* result,
                                hipDoubleComplex* input1,
                                double input2,
                                int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                result[i].x = input1[i].x * input2;
                result[i].y = input1[i].y * input2;
            }

        }

        /**
         * @brief Extract layers orthogonal to the x-axis from three dimensional arrays
         *
         * \note Differently from solver and simple kernels, it is absolutely indispensable to call this kernel
         * **on a two-dimensional grid of blocks**
         *
         * */

        __global__ void extract_layer_x(double* d_layer,
                                        double* d_array,
                                        int layer_index,
                                        int nx,
                                        int ny,
                                        int nz)
        {
            int j = blockIdx.x * blockDim.x + threadIdx.x;
            int k = blockIdx.y * blockDim.y + threadIdx.y;
            if(j < ny && k < nz)
                d_layer[nz*j+k] = d_array[nz*ny*layer_index+nz*j+k];
        }

        /**
         * @brief Extract layers orthogonal to the y-axis from three dimensional arrays
         *
         * \note Differently from solver and simple kernels, it is absolutely indispensable to call this kernel
         * **on a two-dimensional grid of blocks**
         *
         * */

        __global__ void extract_layer_y(double* d_layer,
                                        double* d_array,
                                        int layer_index,
                                        int nx,
                                        int ny,
                                        int nz)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int k = blockIdx.y * blockDim.y + threadIdx.y;
            if(i < nx && k < nz)
                d_layer[nz*i+k] = d_array[nz*ny*i+nz*layer_index+k];
        }

        /**
         * @brief Extract layers orthogonal to the z-axis from three dimensional arrays
         *
         * \note Differently from solver and simple kernels, it is absolutely indispensable to call this kernel
         * **on a two-dimensional grid of blocks**
         *
         * */

        __global__ void extract_layer_z(double* d_layer,
                                        double* d_array,
                                        int layer_index,
                                        int nx,
                                        int ny,
                                        int nz)
        {
            int i = blockIdx.x * blockDim.x + threadIdx.x;
            int j = blockIdx.y * blockDim.y + threadIdx.y;
            if(i < nx && j < ny)
                d_layer[ny*i+j] = d_array[nz*ny*i+nz*j+layer_index];
        }

        /**
         * @brief Low pass filter, kill all momenta above a certain threshold
         * @param
         * */

        __global__ void low_pass_filter(hipDoubleComplex* ft_filtered,
                                        hipDoubleComplex* ft_to_filter,
                                        double* kmod2,
                                        double momentum_cutoff,
                                        int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if(index<size && sqrt(kmod2[index]) < momentum_cutoff)
                ft_filtered[index] = ft_to_filter[index];
        }

        /**
         * @brief Apply a density threshold on a known wave function
         *
         * */

        __global__ void density_threshold(hipDoubleComplex* psi,
                                          double* density,
                                          double n_threshold,
                                          int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            if(index<size)
            {
                double current_density = psi[index].x*psi[index].x+psi[index].y*psi[index].y;
                density[index] = 0.0;
                if(current_density < n_threshold)
                    density[index] = 1.0;
            }
        }
    }

}
