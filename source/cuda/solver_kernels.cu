#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
*
*    This file is part of the UltraCold project.
*
*    UltraCold is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    any later version.
*    UltraCold is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*    You should have received a copy of the GNU General Public License
*    along with UltraCold.  If not, see <https://www.gnu.org/licenses/>.
*
*--------------------------------------------------------------------------------*/

#include "solver_kernels.cuh"
#include "simple_kernels.cuh"

#define PI 3.1415926535897932384626433
#define TWOPI (2*PI)

namespace UltraCold
{
    namespace SolverKernels
    {
        /**
 *
 * @brief Second step in the operator splitting method for gradient descent
 *
 * */

        __global__ void step_2_hpsi(hipDoubleComplex* hpsi,
                                    hipDoubleComplex* psi,
                                    double* Vext,
                                    double* scattering_length,
                                    int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                hpsi[i].x = hpsi[i].x +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y)
                            ) *
                            psi[i].x;
                hpsi[i].y = hpsi[i].y +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y)
                            ) *
                            psi[i].y;
            }
        }

        /**
         *
         * @brief Second step in the operator splitting method for gradient descent for dipolars
         *
         * */

        __global__ void step_2_dipolar_hpsi(hipDoubleComplex* hpsi,
                                            hipDoubleComplex* psi,
                                            double* Vext,
                                            hipDoubleComplex* Phi_dd,
                                            double* scattering_length,
                                            double* gamma_epsilon_dd,
                                            int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            double aux=0.0;
            for (int i = index; i < size; i += stride)
            {
                aux= hipCabs(psi[i]);
                hpsi[i].x = hpsi[i].x +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*pow(aux,2)
                             + Phi_dd[i].x
                             + gamma_epsilon_dd[0]*pow(aux,3)
                            ) *
                            psi[i].x;
                hpsi[i].y = hpsi[i].y +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*pow(aux,2)
                             + Phi_dd[i].x
                             + gamma_epsilon_dd[0]*pow(aux,3)
                            ) *
                            psi[i].y;
            }
        }

        /**
         *
         * @brief Overload for advective velocity field in second step in the operator splitting method for gradient descent for dipolars
         *
         * */

        __global__ void step_2_dipolar_hpsi(hipDoubleComplex* hpsi,
                                            hipDoubleComplex* psi,
                                            double* Vext,
                                            hipDoubleComplex* Phi_dd,
                                            double* scattering_length,
                                            hipDoubleComplex* velocity_x_d,
                                            hipDoubleComplex* velocity_y_d,
                                            hipDoubleComplex* grad_psi_x_d,
                                            hipDoubleComplex* grad_psi_y_d,
                                            double* gamma_epsilon_dd,
                                            int size)
        {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            double aux=0.0;
            for (int i = index; i < size; i += stride)
            {
                aux= hipCabs(psi[i]);
                hpsi[i].x = hpsi[i].x +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*pow(aux,2)
                             + Phi_dd[i].x
                             + gamma_epsilon_dd[0]*pow(aux,3)
                             + pow(velocity_x_d[i].x,2) / 2
                             + pow(velocity_y_d[i].x,2) / 2
                            ) * psi[i].x
                            - velocity_x_d[i].x*grad_psi_x_d[i].y
                            - velocity_y_d[i].x*grad_psi_y_d[i].y;
                hpsi[i].y = hpsi[i].y +
                            (Vext[i]
                             + 4*PI*scattering_length[0]*pow(aux,2)
                             + Phi_dd[i].x
                             + gamma_epsilon_dd[0]*pow(aux,3)
                             + pow(velocity_x_d[i].x,2) / 2
                             + pow(velocity_y_d[i].x,2) / 2
                            ) *
                            psi[i].y
                            + velocity_x_d[i].x*grad_psi_x_d[i].x
                            + velocity_y_d[i].x*grad_psi_y_d[i].x;
            }
        }


        /**
         * @brief Gradient descent plus heavy-ball step
         *
         * */

        __global__ void gradient_descent_step(hipDoubleComplex* psi,
                                              hipDoubleComplex* hpsi,
                                              hipDoubleComplex* psi_new,
                                              hipDoubleComplex* psi_old,
                                              double* alpha,
                                              double* beta,
                                              int size)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                psi_new[i].x = (1.0 + beta[0])*psi[i].x - alpha[0]*hpsi[i].x - beta[0]*psi_old[i].x;
                psi_old[i].x = psi[i].x;
                psi_new[i].y = (1.0 + beta[0])*psi[i].y - alpha[0]*hpsi[i].y - beta[0]*psi_old[i].y;
                psi_old[i].y = psi[i].y;
            }
        }

        /**
         * @brief Solve step-1 operator splitting
         *
         * */

        __global__ void step_1_operator_splitting(hipDoubleComplex* psi,
                                                  double* Vext,
                                                  double* time_step,
                                                  double* scattering_length,
                                                  int size)
        {
            hipDoubleComplex aux;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux.x = 0.0;
                aux.y = - time_step[0] * (Vext[i] + 4*PI*scattering_length[0]*(psi[i].x*psi[i].x+psi[i].y*psi[i].y) );
                psi[i] = hipCmul(psi[i],SimpleKernels::complex_exponential(aux));
            }
        }

        /**
         * @brief Solve step-1 operator splitting for dipolars
         *
         * */

        __global__ void step_1_operator_splitting_dipolars(hipDoubleComplex* psi,
                                                           double* Vext,
                                                           hipDoubleComplex* Phi_dd,
                                                           double* time_step,
                                                           double* scattering_length,
                                                           double* gamma_epsilon_dd,
                                                           int size)
        {
            hipDoubleComplex aux;
            double aux2;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux2 = hipCabs(psi[i]);
                aux.x = 0.0;
                aux.y = - time_step[0] * (Vext[i]
                                          + 4*PI*scattering_length[0]*pow(aux2,2)
                                          + Phi_dd[i].x
                                          + gamma_epsilon_dd[0]*pow(aux2,3)
                );
                psi[i] = hipCmul(psi[i],SimpleKernels::complex_exponential(aux));
            }
        }

        /**
         * @brief Solve step-1 operator splitting for dipolars - driven dissipative
         *
         * */

        __global__ void step_1_operator_splitting_dipolars(hipDoubleComplex* psi,
                                                           double* Vext,
                                                           hipDoubleComplex* Phi_dd,
                                                           double* time_step,
                                                           double* scattering_length,
                                                           double* gamma_epsilon_dd,
                                                           double* gamma_diss,
                                                           int size)
        {
            hipDoubleComplex aux;
            double aux2;
            double aux3;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux2 = hipCabs(psi[i]);
                aux3 = Vext[i] + 4*PI*scattering_length[0]*pow(aux2,2)
                       + Phi_dd[i].x + gamma_epsilon_dd[0]*pow(aux2,3);
                aux.x = - time_step[0] * gamma_diss[0] * aux3;
                aux.y = - time_step[0] * aux3;
                psi[i] = hipCmul(psi[i],SimpleKernels::complex_exponential(aux));
            }
        }

        /**
         *
         * @brief A useful help for step-2 of operator splitting
         *
         * */

        __global__ void aux_step_2_operator_splitting(hipDoubleComplex* psitilde,
                                                      double* kmod2,
                                                      double* time_step,
                                                      int size)
        {
            hipDoubleComplex aux;
            aux.x = 0.0;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux.y = - 0.5 * time_step[0] * pow(TWOPI,2) * kmod2[i];
                psitilde[i] = hipCmul(psitilde[i],SimpleKernels::complex_exponential(aux));
            }
        }

        /**
         *
         * @brief A useful help for step-2 of operator splitting - driven dissipative
         *
         * */

        __global__ void aux_step_2_operator_splitting(hipDoubleComplex* psitilde,
                                                      double* kmod2,
                                                      double* time_step,
                                                      double* gamma_diss,
                                                      int size)
        {
            hipDoubleComplex aux;
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int i = index; i < size; i += stride)
            {
                aux.x = - 0.5 * time_step[0] * gamma_diss[0] * pow(TWOPI,2) * kmod2[i];
                aux.y = - 0.5 * time_step[0] * pow(TWOPI,2) * kmod2[i];
                psitilde[i] = hipCmul(psitilde[i],SimpleKernels::complex_exponential(aux));
            }
        }
    }
}